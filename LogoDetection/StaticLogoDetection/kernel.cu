#include "hip/hip_runtime.h"
#include "device_atomic_functions.hpp"
#include "device_functions.hpp"
#include "hip/hip_runtime.h"
#include ""
#include "ImgProcess.h"
#include <stdio.h>
#define Pi  3.14159265359
__global__  void general2final_kernel(int iw, int ih, float *source, unsigned char *dest)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	dest[iw*y + x] = (unsigned char)source[iw*y + x];
}
__global__  void treshold_kernel(int iw, int ih, int binary_treshold, unsigned char *source, unsigned char *dest)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;

	if (y < 10)
		dest[iw*y + x] = 0;
	if (y > ih - 10)
		dest[iw*y + x] = 0;
	if (x < 10)
		dest[iw*y + x] = 0;
	if (x > iw - 10)
		dest[iw*y + x] = 0;

	if ((unsigned char)dest[iw*y + x] > 60)
		dest[iw*y + x] = 255;
	else
		dest[iw*y + x] = 0;


	__syncthreads();

}
__global__  void Profile_kernel(int iw, int ih, unsigned char *source, double *dest)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if (x >= 0 && x < iw  && y >= 0 && y < ih)
	{
		dest[x] += source[iw*y + x];
		//atomicAdd(&dest[x], source[iw*y + x]); // so better but I dont know why it doesnt declared :| 
	}


}

__global__ void sinc_kernel(int iw, int ih, double a1, double a2, unsigned char *source, unsigned char *dest)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	int Landa = 800, Betta = 100;
	double brightness;
	int offset_cols = iw;
	int offset_rows = ih;
	double P_a1 = a1*a1;
	double P_a2 = a2*a2;
	brightness = -1 * Landa*sin(Pi*pow(P_a1*(x - offset_cols / 2)*(x - offset_cols / 2)*1.0 + P_a2*(y - offset_rows / 2)*(y - offset_rows / 2)*1.0, 0.5)) / (Pi*pow(P_a1*(x - offset_cols / 2)*(x - offset_cols / 2)*1.0 + P_a2*(y - offset_rows / 2)*(y - offset_rows / 2)*1.0, 0.5)) + Betta; // Y must be more than X in rectangular image when cols is more than rows

	if (brightness > 255)
		brightness = 255;
	if (brightness < 0)

		brightness = 0;
	if (brightness < 50)
		dest[iw*y + x] = (unsigned char)brightness;
}
__global__ void generalgradient_kernel(int iw, int ih, int frameCount, unsigned char *source, float *dest)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	float temp;
	if (x > 0 && x < iw - 1 && y > 0 && y < ih - 1)
	{
		temp = dest[iw*y + x];
		dest[iw*y + x] = (float)(1.0*((frameCount - 1)*temp + source[iw*y + x]) / frameCount);
	}

}


__global__ void boxfilter_kernel(int iw, int ih, unsigned char *source, unsigned char *dest, int bw, int bh)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;

	int count = 0;
	float sum = 0.0;

	for (int j = -(bh / 2); j <= (bh / 2); j++)
		for (int i = -(bw / 2); i <= (bw / 2); i++)
		{
			if ((x + i) < iw && (x + i) >= 0 && (y + j) < ih && (y + j) >= 0)
			{
				sum += (float)source[((y + j)*iw) + (x + i)];
				count++;
			}
		}
	sum /= (float)count * 2;
	dest[(y*iw) + x] = (unsigned char)sum;
}
__global__ void sobelfilter_kernel(int iw, int ih, unsigned char *source, unsigned char *dest)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if (x > 0 && x < iw - 1 && y > 0 && y < ih - 1)
	{
		int gx = -1 * source[iw*(y - 1) + (x - 1)] + source[iw*(y - 1) + (x + 1)] +
			-2 * source[iw*y + (x - 1)] + 2 * source[iw*y + (x + 1)] +
			-1 * source[iw*(y + 1) + (x - 1)] + source[iw*(y + 1) + (x + 1)];
		int gy = -source[iw*(y - 1) + (x - 1)] - 2 * source[iw*(y - 1) + x]
			- source[iw*(y - 1) + (x + 1)] +
			source[iw*(y + 1) + (x - 1)] + 2 * source[iw*(y + 1) + x] +
			source[iw*(y + 1) + (x + 1)];
		dest[iw*y + x] = (unsigned char)sqrt((float)gx*(float)gx + (float)gy*float(gy));

	}
}

extern "C" void boxfilter(int iw, int ih, unsigned char *source, unsigned char *dest, int bw, int bh)
{
	unsigned char *dev_source, *dev_dest;
	hipHostGetDevicePointer(&dev_source, source, 0);
	hipHostGetDevicePointer(&dev_dest, dest, 0);

	dim3 blocks(iw / 16, ih / 16);
	dim3 threads(16, 16);
	boxfilter_kernel << <blocks, threads >> >(iw, ih, dev_source, dev_dest, bw, bh);
	hipDeviceSynchronize();
}
extern "C" void sobelfilter(int iw, int ih, unsigned char *source, unsigned char *dest)
{
	// allocate memory for bitmap 
	unsigned char *dev_source, *dev_dest;
	hipHostGetDevicePointer(&dev_source, source, 0);
	hipHostGetDevicePointer(&dev_dest, dest, 0);
	dim3	block(iw / 16, ih / 16);
	dim3	threads(16, 16);
	sobelfilter_kernel << <block, threads >> >(iw, ih, dev_source, dev_dest);
	hipDeviceSynchronize();

}
extern "C" unsigned char* createImageBuffer(unsigned int bytes)
{
	unsigned char *ptr = NULL;
	hipSetDeviceFlags(hipDeviceMapHost);
	hipHostAlloc(&ptr, bytes, hipHostMallocMapped);
	return ptr;

}
extern "C" float * createImageBufferFloat(unsigned int Bytes)
{
	float *ptr = NULL;
	hipSetDeviceFlags(hipDeviceMapHost);
	hipHostAlloc(&ptr, Bytes, hipHostMallocMapped);
	return ptr;

}
void desetroyImageBuffer(unsigned char* bytes)
{
	hipHostFree(bytes);
}
void sinc(int iw, int ih, double a1, double a2, unsigned char *source, unsigned char *dest)
{
	unsigned char *dev_source, *dev_dest;
	hipHostGetDevicePointer(&dev_source, source, 0);
	hipHostGetDevicePointer(&dev_dest, dest, 0);

	dim3 blocks(iw / 16, ih / 16);
	dim3 threads(16, 16);
	sinc_kernel << <blocks, threads >> >(iw, ih, a1, a2, dev_source, dev_dest);
	hipDeviceSynchronize();
}
void generalgradient(int iw, int ih, int frameCount, unsigned char *source, float  *dest)
{
	unsigned char *dev_source;
	float *dev_dest;
	hipHostGetDevicePointer(&dev_source, source, 0);
	hipHostGetDevicePointer(&dev_dest, dest, 0);

	dim3 blocks(iw / 16, ih / 16);
	dim3 threads(16, 16);
	generalgradient_kernel << <blocks, threads >> >(iw, ih, frameCount, dev_source, dev_dest);
	hipDeviceSynchronize();
}
extern "C" void treshold(int iw, int ih, int binary_treshold, unsigned char *source, unsigned char *dest)
{
	unsigned char *dev_source, *dev_dest;
	hipHostGetDevicePointer(&dev_source, source, 0);
	hipHostGetDevicePointer(&dev_dest, dest, 0);

	dim3 blocks(iw / 16, ih / 16);
	dim3 threads(16, 16);
	treshold_kernel << <blocks, threads >> >(iw, ih, binary_treshold, dev_source, dev_dest);
	hipDeviceSynchronize();
}
extern "C" void profile(int iw, int ih, unsigned char  *img, double *myarray)
{
	unsigned char *dev_source;
	double *dev_dest;
	hipHostGetDevicePointer(&dev_source, img, 0);
	hipHostGetDevicePointer(&dev_dest, myarray, 0);

	dim3 blocks(iw / 16, ih / 16);
	dim3 threads(16, 16);
	Profile_kernel << <blocks, threads >> >(iw, ih, dev_source, dev_dest);
	hipDeviceSynchronize();
}
extern "C" double * createdouble(double Bytes)
{
	double *ptr = NULL;
	hipSetDeviceFlags(hipDeviceMapHost);
	hipHostAlloc(&ptr, Bytes, hipHostMallocMapped);
	return ptr;

}
extern "C" void general2final(int iw, int ih, float  *source, unsigned char *dest)
{
	float *dev_source;
	unsigned char *dev_dest;
	hipHostGetDevicePointer(&dev_source, source, 0);
	hipHostGetDevicePointer(&dev_dest, dest, 0);

	dim3 blocks(iw / 16, ih / 16);
	dim3 threads(16, 16);
	general2final_kernel << <blocks, threads >> >(iw, ih, dev_source, dev_dest);
	hipDeviceSynchronize();
}


